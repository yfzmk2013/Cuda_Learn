//
// Created by yanhao on 17-11-21.
//


#include <stdio.h>
#include <stdlib.h>
//#include "utils.h"
#include <iostream>


//#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include "mul_cublas.h"
// Helper function for using CUDA to add vectors in parallel.
hipblasStatus_t addWithCuda(const hipblasHandle_t &handle,float *c, const float *a, const float *b, unsigned int WA, unsigned int HA, unsigned int WB,
                        unsigned int HB) {

    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;
    hipblasStatus_t hipblasStatus_t;


    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void **) &dev_c, HA * WB * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        // Error;
    }

    cudaStatus = hipMalloc((void **) &dev_a, HA * WA * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus = hipMalloc((void **) &dev_b, HB * WB * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    hipblasSetVector(HA*WA,sizeof(float),a,1,dev_a,1);
    hipblasSetVector(HB*WB,sizeof(float),b,1,dev_b,1);
    // 同步函数
    hipDeviceSynchronize();

    float alpha = 1.0;
    float beta = 0.0;
    hipblasStatus_t = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,WA, HA, WB, &alpha, dev_b, HA, dev_a, HA, &beta, dev_c, HA);

    hipDeviceSynchronize();
    hipblasGetVector(HA*WB,sizeof(float),c,1,dev_c,1);
    //Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return hipblasStatus_t;
}

hipblasStatus_t addWithCuda2(const hipblasHandle_t &handle,float *dev_c, const float *dev_a, const float *dev_b, unsigned int WA, unsigned int HA, unsigned int WB,
                            unsigned int HB){

    float alpha = 1.0;
    float beta = 0.0;
    hipblasStatus_t hipblasStatus_t = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,WA, HA, WB, &alpha, dev_b, HA, dev_a, HA, &beta, dev_c, HA);


}