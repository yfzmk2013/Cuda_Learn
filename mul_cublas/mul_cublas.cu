#include "hip/hip_runtime.h"
//
// Created by yanhao on 17-11-21.
//


#include <stdio.h>
#include <stdlib.h>
//#include "utils.h"
#include <iostream>


//#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include "mul_cublas.h"

// Helper function for using CUDA to add vectors in parallel.
//hipblasStatus_t
//addWithCuda(const hipblasHandle_t &handle, float *c, const float *a, const float *b, unsigned int WA, unsigned int HA,
//            unsigned int WB,
//            unsigned int HB) {
//
//    float *dev_a = 0;
//    float *dev_b = 0;
//    float *dev_c = 0;
//    hipError_t cudaStatus;
//    hipblasStatus_t hipblasStatus_t;
//
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void **) &dev_c, HA * WB * sizeof(float));
//    if (cudaStatus != hipSuccess) {
//        printf( "hipMalloc failed!");
//        // Error;
//    }
//
//    cudaStatus = hipMalloc((void **) &dev_a, HA * WA * sizeof(float));
//    if (cudaStatus != hipSuccess) {
//        printf( "hipMalloc failed!");
//        //goto Error;
//    }
//
//    cudaStatus = hipMalloc((void **) &dev_b, HB * WB * sizeof(float));
//    if (cudaStatus != hipSuccess) {
//        printf( "hipMalloc failed!");
//        //goto Error;
//    }
//
//    hipblasSetVector(HA * WA, sizeof(float), a, 1, dev_a, 1);
//    hipblasSetVector(HB * WB, sizeof(float), b, 1, dev_b, 1);
//    // 同步函数
//    hipDeviceSynchronize();
//
//    float alpha = 1.0;
//    float beta = 0.0;
//    hipblasStatus_t = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, WA, HA, WB, &alpha, dev_b, HA, dev_a, HA, &beta, dev_c,
//                               HA);
//
//    hipDeviceSynchronize();
//    hipblasGetVector(HA * WB, sizeof(float), c, 1, dev_c, 1);
//    //Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    return hipblasStatus_t;
//}

//hipblasStatus_t
//addWithCuda(const hipblasHandle_t &handle, float *c, const float *a, const float *b, unsigned int WA, unsigned int HA,
//            unsigned int WB,
//            unsigned int HB) {
//
//    float *dev_a = 0;
//    float *dev_b = 0;
//    float *dev_c = 0;
//    hipError_t cudaStatus;
//    hipblasStatus_t hipblasStatus_t;
//
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void **) &dev_c, HA * WB * sizeof(float));
//    if (cudaStatus != hipSuccess) {
//        printf( "hipMalloc failed!");
//        // Error;
//    }
//
//    cudaStatus = hipMalloc((void **) &dev_a, HA * WA * sizeof(float));
//    if (cudaStatus != hipSuccess) {
//        printf( "hipMalloc failed!");
//        //goto Error;
//    }
//
//    cudaStatus = hipMalloc((void **) &dev_b, HB * WB * sizeof(float));
//    if (cudaStatus != hipSuccess) {
//        printf( "hipMalloc failed!");
//        //goto Error;
//    }
//
//    hipblasSetVector(HA * WA, sizeof(float), a, 1, dev_a, 1);
//    hipblasSetVector(HB * WB, sizeof(float), b, 1, dev_b, 1);
//    // 同步函数
//    hipDeviceSynchronize();
//
//    float alpha = 1.0;
//    float beta = 0.0;
//    clock_t start = clock();
//
//    hipblasStatus_t = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, WA, HA, WB, &alpha, dev_b, HA, dev_a, HA, &beta, dev_c,
//                               HA);
//
//
//    clock_t time_used = clock() - start;
//    printf("(GPU31) time:%ld\n", time_used);
//    hipDeviceSynchronize();
//    hipblasGetVector(HA * WB, sizeof(float), c, 1, dev_c, 1);
//    for (int i = 0; i < 2; ++i) {
//        for (int j = 0; j < 2; ++j) {
//            printf("%f\n", c[i * 2 + j]);
//        }
//    }
//    //Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    return hipblasStatus_t;
//}


//addWithCuda2(const hipblasHandle_t &handle, float *dev_c, const float *dev_a, const float *dev_b, unsigned int WA,
//             unsigned int HA, unsigned int WB,
//             unsigned int HB) {
//
//    float alpha = 1.0;
//    float beta = 0.0;
//    hipblasStatus_t hipblasStatus_t = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, WA, HA, WB, &alpha, dev_b, HA, dev_a,
//                                              HA, &beta, dev_c, HA);
//
//
//}

hipblasStatus_t addWithCuda2(const hipblasHandle_t &handle,float *c, const float *a, const float *b, unsigned int WA, unsigned int HA, unsigned int WB,
                            unsigned int HB) {

    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;
    hipblasStatus_t hipblasStatus_t;


    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void **) &dev_c, HA * WB * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        // Error;
    }

    cudaStatus = hipMalloc((void **) &dev_a, HA * WA * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    cudaStatus = hipMalloc((void **) &dev_b, HB * WB * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        //goto Error;
    }

    hipblasSetVector(HA * WA, sizeof(float), a, 1, dev_a, 1);
    hipblasSetVector(HB * WB, sizeof(float), b, 1, dev_b, 1);
    // 同步函数
    hipDeviceSynchronize();

    float alpha = 1.0;
    float beta = 0.0;
    clock_t start = clock();

    hipblasStatus_t = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, WA, HA, WB, &alpha, dev_b, HA, dev_a, HA, &beta, dev_c,
                               HA);

    hipDeviceSynchronize();

    clock_t time_used = clock() - start;
    printf("(GPU31) time:%ld\n", time_used);
    hipDeviceSynchronize();
    hipblasGetVector(HA * WB, sizeof(float), dev_c, 1,c, 1);
    //Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return hipblasStatus_t;
}

template< typename T>
void gpu_memory_alloc(size_t len, T * &ptr)
{
    hipMalloc(&ptr, sizeof(T) * len);
}
void run(const hipblasHandle_t &handle,const hipStream_t&stream,float *a,float*b,float *c)
{


    float *d_a , *d_b, *d_c;



    gpu_memory_alloc<float>(6, d_a);
    gpu_memory_alloc<float>(8, d_b);
    gpu_memory_alloc<float>(12, d_c);

    hipMemcpy(d_a, a, sizeof(float)* 6, hipMemcpyDefault);
    hipMemcpy(d_b, b, sizeof(float)* 8, hipMemcpyDefault);
    float alph = 1.0f;
    float beta = 0.0f;

    /// a(3*2)    b(2 *4 )
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 3, 2, &alph, d_b, 4, d_a, 2 ,&beta, d_c, 4 );
    hipMemcpyAsync(c, d_c, 12 * sizeof(float), hipMemcpyDefault, stream);

    hipStreamSynchronize(stream);
    printf("aaaaaaaaaaaaaaaaaaaaaa!!!\n");


}