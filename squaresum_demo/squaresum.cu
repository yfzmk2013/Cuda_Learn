#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include "utils.h"
#include <iostream>
#include "squaresum.h"
// ======== define area ========
#define DATA_SIZE 1048576 // 1M

// ======== global area ========
int data[DATA_SIZE];

__global__ static void squaresSum(int *data, int *sum, clock_t *time)
{
    int sum_t = 0;
    clock_t start = clock();
    for (int i = 0; i < DATA_SIZE; ++i) {
        sum_t += data[i] * data[i];
    }
    *sum = sum_t;
    *time = clock() - start;
}

// ======== used to generate rand datas ========
void generateData(int *data, int size)
{
    for (int i = 0; i < size; ++i) {
        data[i] = rand() % 10;
    }
}

int squaresum()
{
    // init CUDA device
    if (!InitCUDA()) {
        return 0;
    }
    printf("CUDA initialized.\n");

    // generate rand datas
    generateData(data, DATA_SIZE);

    // malloc space for datas in GPU
    int *gpuData, *sum;
    clock_t *time;
    hipMalloc((void**) &gpuData, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &sum, sizeof(int));
    hipMalloc((void**) &time, sizeof(clock_t));
    hipMemcpy(gpuData, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    // calculate the squares‘s sum
    squaresSum<<<1, 1, 0>>>(gpuData, sum, time);

    // copy the result from GPU to HOST
    int result;
    clock_t time_used;
    hipMemcpy(&result, sum, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);

    // free GPU spaces
    hipFree(gpuData);
    hipFree(sum);
    hipFree(time);

    // print result
    printf("(GPU) sum:%d time:%ld\n", result, time_used);

    // CPU calculate
    result = 0;
    clock_t start = clock();
    for (int i = 0; i < DATA_SIZE; ++i) {
        result += data[i] * data[i];
    }
    time_used = clock() - start;
    printf("(CPU) sum:%d time:%ld\n", result, time_used);

    return 0;
}