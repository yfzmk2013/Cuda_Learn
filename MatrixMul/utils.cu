#include "utils.h"
#include <hip/hip_runtime.h>
#include <iostream>

void printDeviceProp(const hipDeviceProp_t &prop) {
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA() {
    //used to count the device numbers
    int count;

    // get the cuda device count
    hipGetDeviceCount(&count);
// print("%d\n", count);
    std::cout << count << std::endl;
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    // find the device >= 1.X
    int i = 0;
    for (i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                printDeviceProp(prop);
                break;
            }
        }
    }

    // if can‘t find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    // set cuda device
    hipSetDevice(i);

    return true;
}

//int main(){
//  InitCUDA();
//}