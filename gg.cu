#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//#include <Windows.h>
#include <string.h>
#include <malloc.h>
#include "opencv2/opencv.hpp"
#include "hip/device_functions.h"
__global__ void MatrixMulGPU_1(float *c, const float *a, const float *b, unsigned int WA, unsigned int WB) {
    float sum = 0;
    //找出该线程所在的行和列
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    //线程Thread(row, col)负责计算C(row, col)
    for (int i = 0; i < WB; ++i) {
        sum += a[row * WA + i] * b[i * WB + col];
    }

    c[row * WB + col] = sum;
}

template<int BLOCK_SIZE>
__global__ void MatrixMulGPU_2(float *c, const float *a, const float *b, unsigned int WA, unsigned int WB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = WA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + WA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * WB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int i = aBegin, j = bBegin;
         i <= aEnd;
         i += aStep, j += bStep) {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = a[i + WA * ty + tx];
        Bs[ty][tx] = b[j + WB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        //__syncthreads();
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int k = WB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    c[k + WB * ty + tx] = Csub;
}